#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define DEV_NO 0

/*
// 不要用半精度，gtx沒完全support
// from cuda_runtime.h
#define make_int4(a, b, c, d) ((int4){.x = a, .y = b, .z = c, .w = d})

// from helper_math.h
inline __host__ __device__ int4 operator+(int4 a, int4 b){
    return make_int4(a.x + b.x, a.y + b.y, a.z + b.z,  a.w + b.w);
}
inline __host__ __device__ int4 min(int4 a, int4 b)
{
    return make_int4(min(a.x,b.x), min(a.y,b.y), min(a.z,b.z), min(a.w,b.w));
}
*/

const int INF = ((1 << 30) - 1);
const int B = 64;
const int n_thread = 32;
const int n_per_iter = 4;
const int n_sm_size = 4096;
const int shift = 6;

void input(char* infile);
void output(char* outFileName);
void block_FW();

__global__ void cal1(int *D, int r, int n);
__global__ void cal2(int *D, int r, int n);
__global__ void cal3(int *D, int r, int n);

int n, m, padding_n;
int *Dist, *dev_Dist;
size_t Dist_size;

int main(int argc, char* argv[]){

    input(argv[1]);

    block_FW();
    output(argv[2]);
    return 0;
}

void input(char* infile){
    FILE* file = fopen(infile, "rb");
    fread(&n, sizeof(int), 1, file);
    fread(&m, sizeof(int), 1, file);

    padding_n = n + B -((n % B + B - 1) % B + 1);
    Dist_size = padding_n * padding_n * sizeof(int);
    hipHostMalloc(&Dist, Dist_size, hipHostMallocDefault);
    for(int i = 0; i < padding_n; i++){
        for(int j = 0; j < padding_n; j++){
            if((i == j) && (i < n)) {
                Dist[i * padding_n + j] = 0;
            }else{
                Dist[i * padding_n + j] = INF;
            }
        }
    }

    int pair[3];
    for(int i = 0; i < m; i++){
        fread(pair, sizeof(int), 3, file);
        Dist[pair[0] * padding_n + pair[1]] = pair[2];
    }
    fclose(file);
}

void output(char* outFileName){
    FILE* outfile = fopen(outFileName, "w");
    for(int i = 0; i < n; i++){
        fwrite(&Dist[i * padding_n], sizeof(int), n, outfile);
    }
    fclose(outfile);
}

void block_FW(){
    int round = (padding_n + B - 1) / B;
    dim3 block(n_thread, n_thread), grid2(2, round - 1), grid3(round - 1, round - 1);
    hipMalloc(&dev_Dist, Dist_size);
    hipMemcpy(dev_Dist, Dist, Dist_size, hipMemcpyHostToDevice);

    for(int r = 0; r < round; r++){
        cal1 <<<1, block>>> (dev_Dist, r, padding_n);
        cal2 <<<grid2, block>>> (dev_Dist, r, padding_n);
        cal3 <<<grid3, block>>> (dev_Dist, r, padding_n);
    }
    
    hipMemcpy(Dist, dev_Dist, Dist_size, hipMemcpyDeviceToHost);
    //cudaFree(dev_Dist);
}

__global__ void cal1(int *D, int r, int n){
    __shared__ int sm[n_sm_size]; // 64 * 64

    // local
    int b_i, b_j;
    int i, j;
    int ij, ik, kj;
    
    // 只有pivot
    b_i = b_j = r << shift; 
    i = threadIdx.y, j = threadIdx.x; // < n_thread
    ij = (threadIdx.y * n_thread + threadIdx.x) * n_per_iter;

    // memory放一起
    // 先放ij
    sm[ij] = D[(b_i + i) * n + b_j + j]; 
    sm[ij + 1] = D[(b_i + i) * n + b_j + j + n_thread];
    sm[ij + 2] = D[(b_i + i + n_thread) * n + b_j + j];
    sm[ij + 3] = D[(b_i + i + n_thread) * n + b_j + j + n_thread];

    // 再做min(ij, ik + kj)
    #pragma unroll n_thread
    for(int k = 0; k < n_thread; k++){
        __syncthreads(); //牽扯到share memory都要同步化
        ik = (i * n_thread + k) * 4;
        kj = (k * n_thread + j) * 4;
        sm[ij] = min(sm[ij], sm[ik] + sm[kj]);
        sm[ij + 1] = min(sm[ij + 1], sm[ik] + sm[kj + 1]);
        sm[ij + 2] = min(sm[ij + 2], sm[ik + 2] + sm[kj]);
        sm[ij + 3] = min(sm[ij + 3], sm[ik + 2] + sm[kj + 1]);

        sm[ij] = min(sm[ij], sm[ik + 1] + sm[kj + 2]);
        sm[ij + 1] = min(sm[ij + 1], sm[ik + 1] + sm[kj + 3]);
        sm[ij + 2] = min(sm[ij + 2], sm[ik + 3] + sm[kj + 2]);
        sm[ij + 3] = min(sm[ij + 3], sm[ik + 3] + sm[kj + 3]);
    }

    D[(b_i + i) * n + b_j + j] = sm[ij]; 
    D[(b_i + i) * n + b_j + j + n_thread] = sm[ij + 1]; 
    D[(b_i + i + n_thread) * n + b_j + j] = sm[ij + 2]; 
    D[(b_i + i + n_thread) * n + b_j + j + n_thread] = sm[ij + 3]; 
};

__global__ void cal2(int *D, int r, int n){
    __shared__ int sm[n_sm_size]; // 64 * 64
    __shared__ int cp[n_sm_size]; // 兩個都用sm會讓access亂掉
    
    int b_i, b_j, b_k;
    int i, j;
    volatile int ik, kj;
    int tmp[4];
    
    // 不算pivot，在邊上
    b_i = (blockIdx.x * r + (!blockIdx.x) * (blockIdx.y + (blockIdx.y >= r))) << shift;
    b_j = (blockIdx.x * (blockIdx.y + (blockIdx.y >= r)) + (!blockIdx.x) * r) << shift;
    b_k = r << shift;
    i = threadIdx.y, j = threadIdx.x;
    ik = kj = (i * n_thread + j) * n_per_iter;

    // local
    tmp[0] = D[(b_i + i) * n + b_j + j]; 
    tmp[1] = D[(b_i + i) * n + b_j + j + n_thread];
    tmp[2] = D[(b_i + i + n_thread) * n + b_j + j];
    tmp[3] = D[(b_i + i + n_thread) * n + b_j + j + n_thread];

    // ij
    sm[ik] = D[(b_i + i) * n + b_k + j]; 
    sm[ik + 1] = D[(b_i + i) * n + b_k + j + n_thread];
    sm[ik + 2] = D[(b_i + i + n_thread) * n + b_k + j];
    sm[ik + 3] = D[(b_i + i + n_thread) * n + b_k + j + n_thread];

    // kj
    cp[kj] = D[(b_k + i) * n + b_j + j]; 
    cp[kj + 1] = D[(b_k + i) * n + b_j + j + n_thread];
    cp[kj + 2] = D[(b_k + i + n_thread) * n + b_j + j];
    cp[kj + 3] = D[(b_k + i + n_thread) * n + b_j + j + n_thread];

    __syncthreads();

    #pragma unroll n_thread
    for(int k = 0; k < n_thread; k++){
        ik = (i * n_thread + k) * n_per_iter;
        kj = (k * n_thread + j) * n_per_iter;
        tmp[0] = min(min(tmp[0], sm[ik] + cp[kj]), sm[ik + 1] + cp[kj + 2]);
        tmp[1] = min(min(tmp[1], sm[ik] + cp[kj + 1]), sm[ik + 1] + cp[kj + 3]);
        tmp[2] = min(min(tmp[2], sm[ik + 2] + cp[kj]), sm[ik + 3] + cp[kj + 2]);
        tmp[3] = min(min(tmp[3], sm[ik + 2] + cp[kj + 1]), sm[ik + 3] + cp[kj + 3]);
    }

    D[(b_i + i) * n + b_j + j] = tmp[0]; 
    D[(b_i + i) * n + b_j + j + n_thread] = tmp[1]; 
    D[(b_i + i + n_thread) * n + b_j + j] = tmp[2]; 
    D[(b_i + i + n_thread) * n + b_j + j + n_thread] = tmp[3]; 
};

__global__ void cal3(int *D, int r, int n){
    __shared__ int sm[n_sm_size]; 
    __shared__ int cp[n_sm_size]; 
    
    int b_i, b_j, b_k;
    int i, j;
    volatile int ik, kj;
    int tmp[4];

    // 剩下一大塊
    b_i = (blockIdx.x + (blockIdx.x >= r)) << shift;
    b_j = (blockIdx.y + (blockIdx.y >= r)) << shift;
    b_k = r << shift;
    i = threadIdx.y, j = threadIdx.x;
    ik = kj = (i * n_thread + j) * n_per_iter;

    tmp[0] = D[(b_i + i) * n + b_j + j]; 
    tmp[1] = D[(b_i + i) * n + b_j + j + n_thread];
    tmp[2] = D[(b_i + i + n_thread) * n + b_j + j];
    tmp[3] = D[(b_i + i + n_thread) * n + b_j + j + n_thread];

    sm[ik] = D[(b_i + i) * n + b_k + j]; 
    sm[ik + 1] = D[(b_i + i) * n + b_k + j + n_thread];
    sm[ik + 2] = D[(b_i + i + n_thread) * n + b_k + j];
    sm[ik + 3] = D[(b_i + i + n_thread) * n + b_k + j + n_thread];

    cp[kj] = D[(b_k + i) * n + b_j + j]; 
    cp[kj + 1] = D[(b_k + i) * n + b_j + j + n_thread];
    cp[kj + 2] = D[(b_k + i + n_thread) * n + b_j + j];
    cp[kj + 3] = D[(b_k + i + n_thread) * n + b_j + j + n_thread];

    __syncthreads();

    #pragma unroll n_thread
    for(int k = 0; k < n_thread; k++){
        ik = (i * n_thread + k) * n_per_iter;
        kj = (k * n_thread + j) * n_per_iter;
        tmp[0] = min(min(tmp[0], sm[ik] + cp[kj]), sm[ik + 1] + cp[kj + 2]);
        tmp[1] = min(min(tmp[1], sm[ik] + cp[kj + 1]), sm[ik + 1] + cp[kj + 3]);
        tmp[2] = min(min(tmp[2], sm[ik + 2] + cp[kj]), sm[ik + 3] + cp[kj + 2]);
        tmp[3] = min(min(tmp[3], sm[ik + 2] + cp[kj + 1]), sm[ik + 3] + cp[kj + 3]);
    }

    D[(b_i + i) * n + b_j + j] = tmp[0];    
    D[(b_i + i) * n + b_j + j + n_thread] = tmp[1];   
    D[(b_i + i + n_thread) * n + b_j + j] = tmp[2];   
    D[(b_i + i + n_thread) * n + b_j + j + n_thread] = tmp[3];    
};